#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
using namespace std;

// Add A and B vector on the GPU. Results stored into C
__global__
void addKernel(int n, float* A, float* B, float* C)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;

    if(i < n)
        C[i] = A[i] + B[i];
}

// Add A and B vector. Results stored into C
int add(int n, float* h_A, float* h_B, float* h_C)
{
    int size = n*sizeof(float);

    // Allocate memory on device and copy data
    float* d_A;
    hipMalloc((void**)&d_A, size);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    float* d_B;
    hipMalloc((void**)&d_B, size);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    float* d_C;
    hipMalloc((void**)&d_C, size);

    // launch Kernel
    cout << "Running 256 threads on " << ceil(n/256.0f) << " blocks -> " << 256*ceil(n/256.0f) << endl;
    addKernel<<<ceil(n/256.0f),256>>>(n, d_A, d_B, d_C);

    // Transfer results back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}

#define A_VAL   1.0f
#define B_VAL   2.0f
#define C_VAL   3.0f // A_VAL + B_VAL
/**
 * Perform addition operation on 2 vectors A and B using GPU
 * Then verify result
 */
int main(int argc, char* argv[])
{
    int n;
  
    // Check if there are enough command-line arguments
    if (argc != 2) {
        fprintf(stderr, "Usage: %s <vector size>\n", argv[0]);
        return EXIT_FAILURE;
    }

    n = atoi(argv[1]);

    vector<float> h_A(n, 1.0f);
    vector<float> h_B(n, 2.0f);
    vector<float> h_C(n);

    add(n, h_A.data(), h_B.data(), h_C.data());

    for(int i = 0; i < h_C.size(); ++i) {
        if(fabs(h_C[i]-3.0f) > 0.00001f) {
            cout << "Validation Failure! C[" << i << "]: " << h_C[i] << endl;
            return EXIT_FAILURE;
        }
    }

    cout << "The program completed successfully" << endl;

    return 0;
}